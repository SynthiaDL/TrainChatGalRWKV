#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define MIN_VALUE (-1e38)

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C,
                               const F *__restrict__ const _w, const F *__restrict__ const _u, const F *__restrict__ const _k, const F *__restrict__ const _v,
                               const F *__restrict__ const last_state, F *__restrict__ const _y, F *__restrict__ const new_state) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int state_offset = (_b * C + _c)*3;

    F u = _u[_c];
    F w = _w[_c];
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    F *__restrict__ const y = _y + _offset;

    F p, q, o;
    if (last_state == NULL) {
        p = 0, q = 0, o = MIN_VALUE;
    } else {
        p = last_state[state_offset+0];
        q = last_state[state_offset+1];
        o = last_state[state_offset+2];
    }
    // p and q are running sums divided by exp(o) (to avoid overflows)
    for (int i = 0; i < T; i++) {
        const int ii = i * C;

        F no = max(o, u + k[ii]);
        F A = exp(o - no);
        F B = exp(u + k[ii] - no);
        y[ii] = (A * p + B * v[ii]) / (A * q + B);

        no = max(w + o, k[ii]);
        A = exp(w + o - no);
        B = exp(k[ii] - no);
        p = A * p + B * v[ii];
        q = A * q + B;
        o = no;
    }
    if (new_state != NULL) {
        new_state[state_offset+0] = p;
        new_state[state_offset+1] = q;
        new_state[state_offset+2] = o;
    }
}

template <typename F>
__global__ void kernel_backward(const int B, const int T, const int C,
                                const F *__restrict__ const _w, const F *__restrict__ const _u, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ const last_state, 
                                const F *__restrict__ const _gy, const F *__restrict__ const gnew_state,
                                F *__restrict__ const _gw, F *__restrict__ const _gu, F *__restrict__ const _gk, F *__restrict__ const _gv, F *__restrict__ const glast_state) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int state_offset  = (_b * C + _c)*3;

    F u = _u[_c];
    F w = _w[_c];
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    const F *__restrict__ const gy = _gy + _offset;

    F *__restrict__ const gk = _gk + _offset;
    F *__restrict__ const gv = _gv + _offset;

    F y[Tmax], z[Tmax], zexp[Tmax];

    F gw = 0, gu = 0;
    F dpdw = 0, dqdw = 0;
    F p, q, o;
    if (last_state == NULL) {
        p = 0, q = 0, o = MIN_VALUE;
    } else {
        p = last_state[state_offset+0];
        q = last_state[state_offset+1];
        o = last_state[state_offset+2];
    }
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        F no = max(o, k[ii] + u);
        F A = exp(o - no);
        F B = exp(k[ii] + u - no);

        F num = A * p + B * v[ii];
        F iden = 1 / (A * q + B);

        y[i] = num * iden;
        z[i] = iden;
        zexp[i] = k[ii] + u - no;

        gw += gy[ii] * (dpdw - dqdw * y[i]) * iden * A;
        gu += gy[ii] * (v[ii] - y[i]) * B * iden;

        no = max(w + o, k[ii]);
        A = exp(w + o - no);
        B = exp(k[ii] - no);
        dpdw = A * (p + dpdw);
        dqdw = A * (q + dqdw);
        p = A * p + B * v[ii];
        q = A * q + B;
        o = no;
    }

    F gp = 0, gq = 0, go = MIN_VALUE;
    if (gnew_state != NULL) {
        gp = gnew_state[state_offset+0];
        gq = gnew_state[state_offset+1];
        go = gnew_state[state_offset+2];
        if (gp == 0 && gq == 0) go = MIN_VALUE;
        gw += (gp * dpdw + gq * dqdw) * exp(o+go);
    }

    for (int i = T - 1; i >= 0; i--) {
        const int ii = i * C;
        F A = gy[ii] * z[i] * exp(zexp[i]);
        F B = exp(k[ii] + go);
        gk[ii] = A * (v[ii] - y[i]) + B * (gp * v[ii] + gq);
        gv[ii] = A + B * gp;

        F no = max(w + go, zexp[i] - k[ii] - u);
        A = exp(w + go - no);
        B = gy[ii] * z[i] * exp(zexp[i] - k[ii] - u - no);
        gp = A * gp + B;
        gq = A * gq - B * y[i];
        go = no;
    }

    // glast_state[2] is not the gradient w.r.t of last_state[2]
    // o (index 2) in last_state is just an exponent for p and q
    // so there are really only 2 elements to differentiate on
    // Similary go (glast_state index 2) is just an exponent for gp and gq
    if (glast_state != NULL) {
        glast_state[state_offset+0] = gp;
        glast_state[state_offset+1] = gq;
        glast_state[state_offset+2] = go;
    }

    // Multiply by w because the w -> -exp(w) preprocessing is halfway in the backwards pass, even though it's not in the forward pass
    const int _offsetBC = _b * C + _c;
    _gw[_offsetBC] += gw * _w[_c];
    _gu[_offsetBC] += gu;
}

void cuda_forward(int B, int T, int C, float *w, float *u, float *k, float *v, float *last_state, float *y, float *new_state) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, last_state, y, new_state);
}

void cuda_backward(int B, int T, int C, float *w, float *u, float *k, float *v, float *last_state, float *gy, float *gnew_state, float *gw, float *gu, float *gk, float *gv, float *glast_state) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_backward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, last_state, gy, gnew_state, gw, gu, gk, gv, glast_state);
}

